#include "hip/hip_runtime.h"
//
// Created by JCW on 2024/11/2.
//
#include "image_loader.cuh"

// Disable strict warnings for this header from the Microsoft Visual C++ compiler.
#ifdef _MSC_VER
    #pragma warning (push, 0)
#endif

#define STB_IMAGE_IMPLEMENTATION
#define STBI_FAILURE_USERMSG
#include "stb_image.h"
#include <utils.hpp>
#include <cstdlib>

__device__ __host__ int image_record::bytes_per_scanline() const {
    return bytes_per_pixel * image_width;
}


image_loader::image_loader() {}

image_loader::image_loader(const char* image_filename) {
    // Loads image data from the specified file.
    // If the image was not loaded successfully, width() and height() will return 0.

    const auto filename = std::string(image_filename);

    if (load("images/" + filename)) return;

    utils::log(std::string("ERROR: Could not load image file '") + image_filename);
}

image_loader::~image_loader() {
    delete[] bdata;
    if (bdata_cuda != nullptr) {
        hipFree(bdata_cuda);
        bdata_cuda = nullptr;
    }
}

bool image_loader::load(const std::string& filename) {
    // Loads the linear (gamma=1) image data from the given file name. Returns true if the
    // load succeeded. The resulting data buffer contains the three [0.0, 1.0]
    // floating-point values for the first pixel (red, then green, then blue). Pixels are
    // contiguous, going left to right for the width of the image, followed by the next row
    // below, for the full height of the image.

    auto n = bytes_per_pixel; // Dummy out parameter: original components per pixel
    fdata = stbi_loadf(filename.c_str(), &image_width, &image_height, &n, bytes_per_pixel);
    if (fdata == nullptr) return false;

    convert_to_bytes();
    STBI_FREE(fdata);
    fdata = nullptr;
    return true;
}

int image_loader::width()  const { return (bdata == nullptr) ? 0 : image_width; }

int image_loader::height() const { return (bdata == nullptr) ? 0 : image_height; }

image_record image_loader::get_record() const {
    image_record record;
    record.image_data = bdata;
    record.image_width = image_width;
    record.image_height = image_height;
    record.bytes_per_pixel = bytes_per_pixel;
    return record;
}

image_record image_loader::get_record_cuda() {
    image_copy_to_cuda();
    image_record record_cuda;
    record_cuda.image_data = bdata_cuda;
    record_cuda.image_width = image_width;
    record_cuda.image_height = image_height;
    record_cuda.bytes_per_pixel = bytes_per_pixel;
    return record_cuda;
}

void image_loader::image_copy_to_cuda() {
    if (bdata_cuda == nullptr) {
        hipMalloc(&bdata_cuda, image_width*image_height*bytes_per_pixel*sizeof(unsigned char));
        hipMemcpy(bdata_cuda,bdata,image_width*image_height*bytes_per_pixel*sizeof(unsigned char),hipMemcpyHostToDevice);
    }
}

int image_loader::clamp(const int x, const int low, const int high) {
    // Return the value clamped to the range [low, high).
    if (x < low) return low;
    if (x < high) return x;
    return high - 1;
}

unsigned char image_loader::float_to_byte(const float value) {
    if (value <= 0.0)
        return 0;
    if (1.0 <= value)
        return 255;
    return static_cast<unsigned char>(256.0 * value);
}

void image_loader::convert_to_bytes() {
    // Convert the linear floating point pixel data to bytes, storing the resulting byte
    // data in the `bdata` member.

    const int total_bytes = image_width * image_height * bytes_per_pixel;
    bdata = new unsigned char[total_bytes];

    // Iterate through all pixel components, converting from [0.0, 1.0] float values to
    // unsigned [0, 255] byte values.

    auto *bptr = bdata;
    auto *fptr = fdata;
    for (auto i=0; i < total_bytes; i++, fptr++, bptr++)
        *bptr = float_to_byte(*fptr);
}

// Restore MSVC compiler warnings
#ifdef _MSC_VER
    #pragma warning (pop)
#endif